#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define CUDA_NUM_THREADS 512
#define CUDA_MAX_NUM_BLOCKS 72

__global__
void ComputeTDF(const int * voxel_grid_occ, float * voxel_grid_TDF, int voxel_grid_dim, int num_occ) {


	int voxel_idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (voxel_idx > (voxel_grid_dim* voxel_grid_dim * voxel_grid_dim))
		return;

	int pt_grid_z = voxel_idx / (voxel_grid_dim * voxel_grid_dim);
	int pt_grid_y = (voxel_idx - (pt_grid_z * voxel_grid_dim * voxel_grid_dim)) / voxel_grid_dim;
	int pt_grid_x = voxel_idx - (pt_grid_z * voxel_grid_dim * voxel_grid_dim) - (pt_grid_y * voxel_grid_dim);

	float _distance = 900;
	for (int i = 0; i < num_occ; i++)
	{
		int x = pt_grid_x - voxel_grid_occ[i * 3 + 0];
		int y = pt_grid_y - voxel_grid_occ[i * 3 + 1];
		int z = pt_grid_z - voxel_grid_occ[i * 3 + 2];
		float temp = (x*x + y*y + z*z);
		if (temp < _distance)
			_distance = temp;
	}
	voxel_grid_TDF[voxel_idx] = _distance;
}

// Helper function for using CUDA to add vectors in parallel.
extern "C"
hipError_t  ComputeTDFWithCuda(const int * voxel_grid_occ, float * voxel_grid_TDF, int voxel_grid_dim, int num_occ)
{

	int * gpu_voxel_grid_occ = 0;
	float * gpu_voxel_grid_TDF = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "111 hipSetDevice failed!  Do you have a CUDA-capable GPU installed? ");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&gpu_voxel_grid_occ, num_occ * 3 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "222 gpu_voxel_grid_occ hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&gpu_voxel_grid_TDF, 27000 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "333 gpu_voxel_grid_TDF hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(gpu_voxel_grid_occ, voxel_grid_occ, num_occ * 3 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "444 gpu_voxel_grid_occ hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(gpu_voxel_grid_TDF, voxel_grid_TDF, 27000 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "555 gpu_voxel_grid_TDF hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	ComputeTDF << <CUDA_MAX_NUM_BLOCKS, CUDA_NUM_THREADS >> >(gpu_voxel_grid_occ, gpu_voxel_grid_TDF, voxel_grid_dim, num_occ);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "666 ComputeTDF launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "777 hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(voxel_grid_TDF, gpu_voxel_grid_TDF, 27000 * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "888 voxel_grid_TDF, gpu_voxel_grid_occ hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(gpu_voxel_grid_occ);
	hipFree(gpu_voxel_grid_TDF);


	return cudaStatus;
}
